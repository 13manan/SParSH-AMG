#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include "iostream"
#include "AMG_gpu_phase_utilities.hpp"

using namespace std;

//Function to pin the memory Locations for Matrix vector and rhs
void pin_AMG_MG_matrix(sp_matrix_mg& A,double* &b,double* &x,sp_matrix_mg& P)
{
    hipError_t S1;
    S1 = hipHostRegister(A.rowptr,(A.nrow+1)*sizeof(int), hipHostRegisterPortable);
    if(S1 != hipSuccess)
    {
        cout<<"Rowpointer unsuccessfully Pinned\n";
    }

    S1 = hipHostRegister(A.colindex,(A.nnz)*sizeof(int), hipHostRegisterPortable);
    if(S1 != hipSuccess)
    {
        cout<<"ColumnIndex unsuccessfully Pinned\n";
    }

    S1 = hipHostRegister(A.val,(A.nnz)*sizeof(double),hipHostRegisterPortable);
    if(S1 != hipSuccess)
    {
        cout<<"Value Array unsuccessfully Pinned\n";
    }

    S1 = hipHostRegister(A.diagonal,(A.nrow)*sizeof(double),hipHostRegisterPortable);
    if(S1 != hipSuccess)
    {
        cout<<"Value Array unsuccessfully Pinned\n";
    }

    S1 = hipHostRegister(b,(A.nrow)*sizeof(double), hipHostRegisterPortable);
    if(S1 != hipSuccess)
    {
        cout<<"rhs unsuccessfully Pinned\n";
    }

    S1 = hipHostRegister(x,(A.nrow)*sizeof(double), hipHostRegisterPortable);
    if(S1 != hipSuccess)
    {
        cout<<"sol unsuccessfully Pinned\n";
    }

    S1 = hipHostRegister(P.rowptr,(P.nrow+1)*sizeof(int), hipHostRegisterPortable);
    if(S1 != hipSuccess)
    {
        cout<<"P Rowpointer unsuccessfully Pinned\n";
    }

    S1 = hipHostRegister(P.colindex,(P.nnz)*sizeof(int), hipHostRegisterPortable);
    if(S1 != hipSuccess)
    {
        cout<<"P ColumnIndex unsuccessfully Pinned\n";
    }

    S1 = hipHostRegister(P.val,(P.nnz)*sizeof(double),hipHostRegisterPortable);
    if(S1 != hipSuccess)
    {
        cout<<"P Value Array unsuccessfully Pinned\n";
    }
}

//Function to unpin the memory Locations for Matrix vector and rhs
void unpin_AMG_MG_matrix(sp_matrix_mg& A,double* &b,double* &x,sp_matrix_mg& P)
{
    hipError_t S1;
    S1 = hipHostUnregister(A.rowptr);
    if(S1 != hipSuccess)
    {
        cout<<"Rowpointer unsuccessfully UnPinned\n";
    }

    S1 = hipHostUnregister(A.colindex);
    if(S1 != hipSuccess)
    {
        cout<<"ColumnIndex unsuccessfully UnPinned\n";
    }

    S1 = hipHostUnregister(A.val);
    if(S1 != hipSuccess)
    {
        cout<<"Value Array unsuccessfully UnPinned\n";
    }

    S1 = hipHostUnregister(A.diagonal);
    if(S1 != hipSuccess)
    {
        cout<<"Value Array unsuccessfully UnPinned\n";
    }

    S1 = hipHostUnregister(b);
    if(S1 != hipSuccess)
    {
        cout<<"rhs Array unsuccessfully UnPinned\n";
    }

    S1 = hipHostUnregister(x);
    if(S1 != hipSuccess)
    {
        cout<<"sol Array unsuccessfully UnPinned\n";
    }

    S1 = hipHostUnregister(P.rowptr);
    if(S1 != hipSuccess)
    {
        cout<<"Rowpointer unsuccessfully UnPinned\n";
    }

    S1 = hipHostUnregister(P.colindex);
    if(S1 != hipSuccess)
    {
        cout<<"ColumnIndex unsuccessfully UnPinned\n";
    }

    S1 = hipHostUnregister(P.val);
    if(S1 != hipSuccess)
    {
        cout<<"Value Array unsuccessfully UnPinned\n";
    }
}

//Function to swap pointers
void gpu_swap_pointers(sp_matrix_gpu* &A1,sp_matrix_gpu* &A2)
{
    sp_matrix_gpu* A3;
    A3 = A1;
    A1 = A2;
    A2 = A3;
}

//Function to compute residual on GPU
double residual(sp_matrix_gpu &A,double* b,double* x,double* h,hipStream_t streams)
{
    A.cusparse_stat = hipsparseSetStream(A.cusparse_hndl,streams);
    double s3 = 0.0;
    double s1 = 1.0;
    A.cusparse_stat = hipsparseDcsrmv(A.cusparse_hndl,A.transA,A.nrow,A.ncol,A.nnz,&s1,A.descrA,A.val,A.rowptr,A.colindex,x,&s3,h);

    if(A.cusparse_stat != HIPSPARSE_STATUS_SUCCESS)
    {
        cout<<"SPMV unSuccessfull"<<endl;
    }
    
    s1 = -1.0;
    A.cublas_stat = hipblasSetStream(A.cublas_hndl,streams);
    A.cublas_stat = hipblasDaxpy(A.cublas_hndl,A.nrow,&s1,b,1,h,1);

    if( A.cublas_stat != HIPBLAS_STATUS_SUCCESS)
    {
        cout<<"b - Ax unSuccessfull"<<endl;
    }

    double result;
    A.cublas_stat = hipblasDnrm2(A.cublas_hndl,A.nrow,h,1,&result);
    if(A.cublas_stat != HIPBLAS_STATUS_SUCCESS)
    {
        cout<<"norm computation unsuccessfull"<<endl;
    }

    return result;
}

