#include "AMG_gpu_phases_2.hpp"
#include "AMG_gpu_phase_utilities.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <thrust/fill.h>
#include <iostream>
#include <thrust/execution_policy.h>
#include <omp.h>

using namespace std;

void AMG_GPU1_solver::GPU_Allocations()
{
    hipError_t S1;

    streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));

    for (int i = 0; i < nstreams; i++)
    {
        S1 = hipStreamCreate(&streams[i]);    
    }

    Avgpu = new sp_matrix_gpu*[l];
    Pvgpu = new sp_matrix_gpu*[l-1];
    
    Xvgpu = new double*[l];
    Bvgpu = new double*[l];
    Rvgpu = new double*[l];

    for(int i = 0;i < l;i++)
    {
        pin_AMG_MG_matrix(*Av[i],Bv[i],Xv[i],*Pv[i]);
    }

    S1 = hipHostRegister(Xv[l],(Av[l]->nrow)*sizeof(double), hipHostRegisterPortable);
    if(S1 != hipSuccess)
    {
        cout<<"Xv[l] unsuccessfully Pinned\n";
    }

    S1 = hipHostRegister(Bv[l],(Av[l]->nrow)*sizeof(double), hipHostRegisterPortable);
    if(S1 != hipSuccess)
    {
        cout<<"Bv[l] unsuccessfully Pinned\n";
    }

    S1 = hipHostRegister(Av[l]->rowptr,(Av[l]->nrow+1)*sizeof(int), hipHostRegisterPortable);
    if(S1 != hipSuccess)
    {
        cout<<"Row ptr unsuccessfully Pinned\n";
    }

    S1 = hipHostRegister(Av[l]->colindex,(Av[l]->nnz)*sizeof(int), hipHostRegisterPortable);
    if(S1 != hipSuccess)
    {
        cout<<"Col Index unsuccessfully Pinned\n";
    }

    S1 = hipHostRegister(Av[l]->val,(Av[l]->nnz)*sizeof(double), hipHostRegisterPortable);
    if(S1 != hipSuccess)
    {
        cout<<"Value unsuccessfully Pinned\n";
    }

    S1 = hipHostRegister(Av[l]->diagonal,(Av[l]->nrow)*sizeof(double), hipHostRegisterPortable);
    if(S1 != hipSuccess)
    {
        cout<<"Diagonal unsuccessfully Pinned\n";
    }

    for(int i = 0;i <= l;i++)
    {
        Avgpu[i] = new sp_matrix_gpu(*Av[i]);
        Avgpu[i]->matrix_transfer_gpu(*Av[i],streams[0]);
        if(i != 0)
        {
            hipMalloc(&(Xvgpu[i]),(Av[i]->nrow)*sizeof(double));
            hipMalloc(&(Bvgpu[i]),(Av[i]->nrow)*sizeof(double));
        }
        
        hipMalloc(&(Rvgpu[i]),(Av[i]->nrow)*sizeof(double));
    }

    for(int i = 0;i < l;i++)
    {
        Pvgpu[i] = new sp_matrix_gpu(*Pv[i]);
        Pvgpu[i]->matrix_transfer_gpu(*Pv[i],streams[0]);
    }

    hipMalloc(&(xgpu),(Av[0]->nrow)*sizeof(double));
    hipMalloc(&(bgpu),(Av[0]->nrow)*sizeof(double));
    
}

void AMG_GPU1_solver::AMG_Solve(double* b,double* x,int iterations)
{
    Xvgpu[0] = x;
    Bvgpu[0] = b;
    int count = 0;
    double s1 = 1.0;
    double s3 = 0.0;
    double s2 = -1.0;

    double r1 = 1.0;

    hipStreamSynchronize(streams[0]);
    hipStreamSynchronize(streams[1]);
    hipDeviceSynchronize();
    if(iterations > 0)
    {
        while(count++ < iterations)
        {
            for(int i = 0;i < l;i++)
            {
                //SMooth 
                Avgpu[i]->smooth_jacobi(Bvgpu[i],Xvgpu[i],Rvgpu[i],streams[0],6);
                
                //Get Residual
                Avgpu[i]->cusparse_stat = hipsparseDcsrmv(Avgpu[i]->cusparse_hndl,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                Avgpu[i]->nrow,Avgpu[i]->ncol,Avgpu[i]->nnz,&s2,Avgpu[i]->descrA,Avgpu[i]->val,Avgpu[i]->rowptr,Avgpu[i]->colindex,Xvgpu[i],&s3,Rvgpu[i]);
                Avgpu[i]->cublas_stat   = hipblasDaxpy(Avgpu[i]->cublas_hndl,Avgpu[i]->nrow,&s1,Bvgpu[i],1,Rvgpu[i],1);

                // Restrict
                Avgpu[i]->cusparse_stat = hipsparseDcsrmv(Avgpu[i]->cusparse_hndl,HIPSPARSE_OPERATION_TRANSPOSE,
                                Pvgpu[i]->nrow,Pvgpu[i]->ncol,Pvgpu[i]->nnz,&s1,Pvgpu[i]->descrA,Pvgpu[i]->val,Pvgpu[i]->rowptr,Pvgpu[i]->colindex,Rvgpu[i],&s3,Bvgpu[i+1]);
                thrust::fill(thrust::device,(Xvgpu[i+1]),(Xvgpu[i+1])+ Av[i+1]->nrow,0.0);
            }

            hipError_t S1;
            S1 = hipMemcpyAsync(Bv[l],Bvgpu[l],(Av[l]->nrow)*sizeof(double),hipMemcpyDeviceToHost,streams[0]);
            if(S1 != hipSuccess)
            {
                cout<<"Error in Bgpu"<<endl;
            }
            hipStreamSynchronize(streams[0]);
            hipStreamSynchronize(streams[1]);
            hipDeviceSynchronize();

            Directsolve->Direct_Solver_Pardiso_solve(Bv[l],Xv[l]);
            S1 = hipMemcpyAsync(Xvgpu[l],Xv[l],(Av[l]->nrow)*sizeof(double),hipMemcpyHostToDevice,streams[0]);

            hipStreamSynchronize(streams[0]);
            hipStreamSynchronize(streams[1]);
            hipDeviceSynchronize();
            for(int i = l-1;i >= 0;i--)
            {
                Avgpu[i]->cusparse_stat = hipsparseDcsrmv(Avgpu[i]->cusparse_hndl,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                Pvgpu[i]->nrow,Pvgpu[i]->ncol,Pvgpu[i]->nnz,&s1,Pvgpu[i]->descrA,Pvgpu[i]->val,Pvgpu[i]->rowptr,Pvgpu[i]->colindex,Xvgpu[i+1],&s1,Xvgpu[i]);
                Avgpu[i]->smooth_jacobi(Bvgpu[i],Xvgpu[i],Rvgpu[i],streams[0],6);
            }

            //Get Residual
            hipStreamSynchronize(streams[0]);
            hipStreamSynchronize(streams[1]);
            hipDeviceSynchronize();

            Avgpu[0]->cusparse_stat = hipsparseDcsrmv(Avgpu[0]->cusparse_hndl,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                Avgpu[0]->nrow,Avgpu[0]->ncol,Avgpu[0]->nnz,&s2,Avgpu[0]->descrA,Avgpu[0]->val,Avgpu[0]->rowptr,Avgpu[0]->colindex,Xvgpu[0],&s3,Rvgpu[0]);
            Avgpu[0]->cublas_stat   = hipblasDaxpy(Avgpu[0]->cublas_hndl,Avgpu[0]->nrow,&s1,Bvgpu[0],1,Rvgpu[0],1);

            
            Avgpu[0]->cublas_stat   = hipblasDnrm2(Avgpu[0]->cublas_hndl,Av[0]->nrow,Rvgpu[0],1,&r1);

            hipStreamSynchronize(streams[0]);
            hipStreamSynchronize(streams[1]);
            hipDeviceSynchronize();
        }
    }

    if(iterations == -1)
    {
        while(r1 > tol1)
        {
            for(int i = 0;i < l;i++)
            {
                //SMooth 
                Avgpu[i]->smooth_jacobi(Bvgpu[i],Xvgpu[i],Rvgpu[i],streams[0],6);
                
                //Get Residual
                Avgpu[i]->cusparse_stat = hipsparseDcsrmv(Avgpu[i]->cusparse_hndl,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                Avgpu[i]->nrow,Avgpu[i]->ncol,Avgpu[i]->nnz,&s2,Avgpu[i]->descrA,Avgpu[i]->val,Avgpu[i]->rowptr,Avgpu[i]->colindex,Xvgpu[i],&s3,Rvgpu[i]);
                Avgpu[i]->cublas_stat   = hipblasDaxpy(Avgpu[i]->cublas_hndl,Avgpu[i]->nrow,&s1,Bvgpu[i],1,Rvgpu[i],1);

                // Restrict
                Avgpu[i]->cusparse_stat = hipsparseDcsrmv(Avgpu[i]->cusparse_hndl,HIPSPARSE_OPERATION_TRANSPOSE,
                                Pvgpu[i]->nrow,Pvgpu[i]->ncol,Pvgpu[i]->nnz,&s1,Pvgpu[i]->descrA,Pvgpu[i]->val,Pvgpu[i]->rowptr,Pvgpu[i]->colindex,Rvgpu[i],&s3,Bvgpu[i+1]);
                thrust::fill(thrust::device,(Xvgpu[i+1]),(Xvgpu[i+1])+ Av[i+1]->nrow,0.0);
            }

            hipError_t S1;
            S1 = hipMemcpyAsync(Bv[l],Bvgpu[l],(Av[l]->nrow)*sizeof(double),hipMemcpyDeviceToHost,streams[0]);
            if(S1 != hipSuccess)
            {
                cout<<"Error in Bgpu"<<endl;
            }

            hipStreamSynchronize(streams[0]);
            hipStreamSynchronize(streams[1]);
            hipDeviceSynchronize();

            Directsolve->Direct_Solver_Pardiso_solve(Bv[l],Xv[l]);
            S1 = hipMemcpyAsync(Xvgpu[l],Xv[l],(Av[l]->nrow)*sizeof(double),hipMemcpyHostToDevice,streams[0]);

            hipStreamSynchronize(streams[0]);
            hipStreamSynchronize(streams[1]);
            hipDeviceSynchronize();
            for(int i = l-1;i >= 0;i--)
            {
                Avgpu[i]->cusparse_stat = hipsparseDcsrmv(Avgpu[i]->cusparse_hndl,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                Pvgpu[i]->nrow,Pvgpu[i]->ncol,Pvgpu[i]->nnz,&s1,Pvgpu[i]->descrA,Pvgpu[i]->val,Pvgpu[i]->rowptr,Pvgpu[i]->colindex,Xvgpu[i+1],&s1,Xvgpu[i]);
                Avgpu[i]->smooth_jacobi(Bvgpu[i],Xvgpu[i],Rvgpu[i],streams[0],6);
            }

            //Get Residual
            hipStreamSynchronize(streams[0]);
            hipStreamSynchronize(streams[1]);
            hipDeviceSynchronize();

            Avgpu[0]->cusparse_stat = hipsparseDcsrmv(Avgpu[0]->cusparse_hndl,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                Avgpu[0]->nrow,Avgpu[0]->ncol,Avgpu[0]->nnz,&s2,Avgpu[0]->descrA,Avgpu[0]->val,Avgpu[0]->rowptr,Avgpu[0]->colindex,Xvgpu[0],&s3,Rvgpu[0]);
            Avgpu[0]->cublas_stat   = hipblasDaxpy(Avgpu[0]->cublas_hndl,Avgpu[0]->nrow,&s1,Bvgpu[0],1,Rvgpu[0],1);
            
            
            
            Avgpu[0]->cublas_stat   = hipblasDnrm2(Avgpu[0]->cublas_hndl,Av[0]->nrow,Rvgpu[0],1,&r1);
            
            if(print_solve_phase_details == 1)
            {
                cout<<count++<<"\t"<<r1<<endl;    
            }
            
            hipStreamSynchronize(streams[0]);
            hipStreamSynchronize(streams[1]);
            hipDeviceSynchronize();
        }
    }
    

}

void AMG_GPU1_solver::helper(double* b,double* x,int iterations)
{

    hipMemcpy(bgpu,b,(Av[0]->nrow)*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(xgpu,x,(Av[0]->nrow)*sizeof(double),hipMemcpyHostToDevice);

    hipStreamSynchronize(streams[0]);
    hipStreamSynchronize(streams[1]);
    hipDeviceSynchronize();

    AMG_Solve(bgpu,xgpu,iterations);

    hipStreamSynchronize(streams[0]);
    hipStreamSynchronize(streams[1]);
    hipDeviceSynchronize();
    
    hipMemcpy(x,xgpu,(Av[0]->nrow)*sizeof(double),hipMemcpyDeviceToHost);
    
    hipStreamSynchronize(streams[0]);
    hipStreamSynchronize(streams[1]);
    hipDeviceSynchronize();
}

AMG_GPU1_solver::~AMG_GPU1_solver()
{
    hipStreamDestroy(streams[0]);
    hipStreamDestroy(streams[1]);
     
    //Unpin Matrices
    int k = 0;
    while(k < l)
    {
        unpin_AMG_MG_matrix(*Av[k],(Bv[k]),(Xv[k]),(*Pv[k]));
        k++;  
    }

    hipError_t S1 = hipHostUnregister(Xv[l]);
    if(S1 != hipSuccess)
    {
        cout<<"Xv[l] unsuccessfully UnPinned\n";
    }

    S1 = hipHostUnregister(Bv[l]);
    if(S1 != hipSuccess)
    {
        cout<<"Bv[l] unsuccessfully UnPinned\n";
    }

    S1 = hipHostUnregister(Av[l]->rowptr);
    if(S1 != hipSuccess)
    {
        cout<<"Row ptr unsuccessfully UnPinned\n";
    }

    S1 = hipHostUnregister(Av[l]->colindex);
    if(S1 != hipSuccess)
    {
        cout<<"Col Index unsuccessfully UnPinned\n";
    }

    S1 = hipHostUnregister(Av[l]->val);
    if(S1 != hipSuccess)
    {
        cout<<"Value unsuccessfully UnPinned\n";
    }

    S1 = hipHostUnregister(Av[l]->diagonal);
    if(S1 != hipSuccess)
    {
        cout<<"diagonal unsuccessfully UnPinned\n";
    }

    
    for(int i = l;i > 0;i--)
    {
        Avgpu[i]->~sp_matrix_gpu();
        Pvgpu[i-1]->~sp_matrix_gpu();

        hipFree(Xvgpu[i]);
        hipFree(Bvgpu[i]);
        hipFree(Rvgpu[i]);
    }

    
    Avgpu[0]->~sp_matrix_gpu();
    hipFree(Rvgpu[0]);

    delete[] Avgpu;
    delete[] Bvgpu;
    delete[] Rvgpu;
    delete[] Xvgpu;
    delete[] Pvgpu;
    
    hipFree(xgpu);
    hipFree(bgpu);

}
