#include "hip/hip_runtime.h"
#include "AMG.hpp"
#include "AMG_gpu_matrix.hpp"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <thrust/fill.h>
#include <thrust/execution_policy.h>
#include <thrust/copy.h>

using namespace std;

//Kernals
__global__ void jacobi_update(double* x,double* h,double* d,int nrow)
{
    int id = thgpu*blockIdx.x + threadIdx.x;
    if(id < nrow)
    {
        x[id] -= omega*h[id]/d[id];
    }
}


//Constructor for GPU matrix object
sp_matrix_gpu::sp_matrix_gpu(sp_matrix_mg &A)
{
    nrow = A.nrow;
    ncol = A.ncol;
    nnz = A.nnz;
    
    hipError_t S1;
   
    S1 = hipMalloc(&rowptr,(A.nrow+1)*sizeof(int));
    if(S1 != hipSuccess)
    {
        cout<<"Rowpointer Allocation Failed\n";
    }
    
    S1 = hipMalloc(&colindex,(A.nnz)*sizeof(int));
    if(S1 != hipSuccess)
    {
        cout<<"Colindex Allocation Failed\n";
    }

    S1 = hipMalloc(&val,(A.nnz)*sizeof(double));
    if(S1 != hipSuccess)
    {
        cout<<"Colindex Allocation Failed\n";
    }

    S1 = hipMalloc(&diag,(A.nrow)*sizeof(double));
    if(S1 != hipSuccess)
    {
        cout<<"Colindex Allocation Failed\n";
    }

    cusparse_stat  = hipsparseCreate(&cusparse_hndl);  
    cublas_stat    = hipblasCreate(&cublas_hndl);

    cusparse_stat = hipsparseCreateMatDescr(&descrA);
    cusparse_stat   = hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
}


//Method to transfer matrix to GPU
void sp_matrix_gpu::matrix_transfer_gpu(sp_matrix_mg &A,hipStream_t streams)
{
    nrow = A.nrow;
    ncol = A.ncol;
    nnz = A.nnz;

    hipError_t S1;
    S1 = hipMemcpyAsync(rowptr,A.rowptr,(A.nrow+1)*sizeof(int), hipMemcpyHostToDevice,streams);
    if(S1 != hipSuccess)
    {
        cout<<"error in 1\n";
    }

    S1 = hipMemcpyAsync(colindex,A.colindex,(A.nnz)*sizeof(int), hipMemcpyHostToDevice,streams);
    if(S1 != hipSuccess)
    {
        cout<<"error in 2\n";
    }

    S1 = hipMemcpyAsync(val,A.val,(A.nnz)*sizeof(double), hipMemcpyHostToDevice,streams);
    if(S1 != hipSuccess)
    {
        cout<<"error in 3\n";
    }
    if(A.nrow == A.ncol)
    {
        S1 = hipMemcpyAsync(diag,A.diagonal,(A.nrow)*sizeof(double), hipMemcpyHostToDevice,streams);
        if(S1 != hipSuccess)
        {
            cout<<"error in 4\n";
        }
    }
    
}


//Method to perform Jacobi iteration on GPU
void sp_matrix_gpu::smooth_jacobi(double *bgpu,double *xgpu,double *hgpu, hipStream_t streams,int steps)
{
    cusparse_stat  = hipsparseSetStream(cusparse_hndl,streams);
    cublas_stat    = hipblasSetStream(cublas_hndl,streams);

    double s1 = 1.0;
    double s2 = -1.0;
    double s3 = 0.0;
    int count = 0;
    int nblock = nrow/thgpu;

    while(count++ < steps)
    {
        cusparse_stat = hipsparseDcsrmv(cusparse_hndl,transA,nrow,ncol,nnz,&s1,descrA,val,rowptr,colindex,xgpu,&s3,hgpu);
        if(cusparse_stat != HIPSPARSE_STATUS_SUCCESS)
        {
            cout<<"SPMV unSuccessfull 1"<<endl;
        }
        cublas_stat = hipblasDaxpy(cublas_hndl,nrow,&s2,bgpu,1,hgpu,1);
        jacobi_update<<<nblock+1,thgpu,0,streams>>>(xgpu,hgpu,diag,nrow);
    };
}

//Method for SOR smoother to be defined

 


//Destructor to free GPU memory
sp_matrix_gpu::~sp_matrix_gpu()
{
    hipFree(rowptr);
    hipFree(colindex);
    hipFree(val);
    hipFree(diag);
}

